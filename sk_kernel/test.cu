#include "hip/hip_runtime.h"
#include "sk_kernel.cuh" 
#include "mask.cuh"

bool float4_arrays_equal(float4 *arr1, float4 *arr2, size_t size) {
    for (size_t i = 0; i < size; i++) {
        if (arr1[i].x != arr2[i].x | arr1[i].y != arr2[i].y | arr1[i].z != arr2[i].z | arr1[i].w != arr2[i].w) {
            printf("first fail at index %lu: %f != %f, %f != %f, %f != %f, %f != %f \n", 
                i, arr1[i].x, arr2[i].x, arr1[i].y, arr2[i].y, arr1[i].z, arr2[i].z, arr1[i].w, arr2[i].w);
            return false;
        }
    }
    return true;
}

bool uint32_arrays_equal(uint32_t *arr1, uint32_t *arr2, size_t size) {
    for (size_t i = 0; i < size; i++) {
        if (arr1[i] != arr2[i]) {
            printf("first fail at index %lu: %lu != %lu \n", i, (unsigned long) arr1[i], (unsigned long) arr2[i]);
            return false;
        }
    }
    return true;
}

bool float_arrays_equal(float *arr1, float *arr2, size_t size) {
    for (size_t i = 0; i < size; i++) {
        if (0.7 > arr1[i] / arr2[i] | arr1[i] / arr2[i] > 1.3) {
            printf("first fail at index %lu: %f != %f \n", i, arr1[i], arr2[i]);
            return false;
        }
    }
    return true;
}

float h_square(float num) {
    return num * num;
}

float h_cmplx_square(float real, float imaginary) {
    return h_square(real) + h_square(imaginary);
}

float h_cmplx_tesseract(float real, float imaginary) { 
    return h_square(h_cmplx_square(real, imaginary));
}

void naive_downsample(uint32_t *E, float4 *S1, float4 *S2, float4 *S1_p, float4 *S2_p, size_t N, size_t N_p, size_t D, size_t T, size_t F) {
    float s1_0, s1_1, s1_2, s1_3, s2_0, s2_1, s2_2, s2_3;
    s1_0 = s1_1 = s1_2 = s1_3 = s2_0 = s2_1 = s2_2 = s2_3 = 0;
    float s1_p_0, s1_p_1, s1_p_2, s1_p_3, s2_p_0, s2_p_1, s2_p_2, s2_p_3;
    s1_p_0 = s1_p_1 = s1_p_2 = s1_p_3 = s2_p_0 = s2_p_1 = s2_p_2 = s2_p_3 = 0;
    float e0_re, e0_im, e1_re, e1_im, e2_re, e2_im, e3_re, e3_im;
    e0_re = e0_im = e1_re = e1_im = e2_re = e2_im = e3_re = e3_im = 0;

    for (int t_bar = 0; t_bar < T; t_bar = t_bar + N_p){
        for (int f = 0; f < F; f++) {
            for (int feed4 = 0; feed4 < D / 2; feed4++) {
                s1_p_0 = 0;
                s2_p_0 = 0;
                s1_p_1 = 0;
                s2_p_1 = 0;
                s1_p_2 = 0;
                s2_p_2 = 0;
                s1_p_3 = 0;
                s2_p_3 = 0;

                for (int t = t_bar; t < t_bar + N_p; t++) {
                    uint32_t e = E[t * D / 2 * F + f * D / 2 + feed4];

                    e0_re = float(e & 0xf);
                    e0_im = float((e >> 4) & 0xf);
                    e1_re = float((e >> 8) & 0xf);
                    e1_im = float((e >> 12) & 0xf);
                    e2_re = float((e >> 16) & 0xf);
                    e2_im = float((e >> 20) & 0xf);
                    e3_re = float((e >> 24) & 0xf);
                    e3_im = float((e >> 28) & 0xf);

                    s1_p_0 += h_cmplx_square(e0_re, e0_im);
                    s1_p_1 += h_cmplx_square(e1_re, e1_im);
                    s1_p_2 += h_cmplx_square(e2_re, e2_im);
                    s1_p_3 += h_cmplx_square(e3_re, e3_im);

                    s2_p_0 += h_cmplx_tesseract(e0_re, e0_im);
                    s2_p_1 += h_cmplx_tesseract(e1_re, e1_im);
                    s2_p_2 += h_cmplx_tesseract(e2_re, e2_im);
                    s2_p_3 += h_cmplx_tesseract(e3_re, e3_im);
                }

                S1_p[t_bar/N_p * D / 2 * F + f * D / 2 + feed4].x = s1_p_0;
                S1_p[t_bar/N_p * D / 2 * F + f * D / 2 + feed4].y = s1_p_1;
                S1_p[t_bar/N_p * D / 2 * F + f * D / 2 + feed4].z = s1_p_2;
                S1_p[t_bar/N_p * D / 2 * F + f * D / 2 + feed4].w = s1_p_3;

                S2_p[t_bar/N_p * D / 2 * F + f * D / 2 + feed4].x = s2_p_0;
                S2_p[t_bar/N_p * D / 2 * F + f * D / 2 + feed4].y = s2_p_1;
                S2_p[t_bar/N_p * D / 2 * F + f * D / 2 + feed4].z = s2_p_2;
                S2_p[t_bar/N_p * D / 2 * F + f * D / 2 + feed4].w = s2_p_3;

            }
        }
    }

    for (int t_bar = 0; t_bar < T; t_bar = t_bar + N){
        for (int f = 0; f < F; f++) {
            for (int feed4 = 0; feed4 < D / 2; feed4++) {
                s1_0 = 0;
                s2_0 = 0;
                s1_1 = 0;
                s2_1 = 0;
                s1_2 = 0;
                s2_2 = 0;
                s1_3 = 0;
                s2_3 = 0;
                
                for (int t = t_bar; t < t_bar + N; t++) {
                    uint32_t e = E[t * D / 2 * F + f * D / 2 + feed4];

                    e0_re = float(e & 0xf);
                    e0_im = float((e >> 4) & 0xf);
                    e1_re = float((e >> 8) & 0xf);
                    e1_im = float((e >> 12) & 0xf);
                    e2_re = float((e >> 16) & 0xf);
                    e2_im = float((e >> 20) & 0xf);
                    e3_re = float((e >> 24) & 0xf);
                    e3_im = float((e >> 28) & 0xf);

                    s1_0 += h_cmplx_square(e0_re, e0_im);
                    s1_1 += h_cmplx_square(e1_re, e1_im);
                    s1_2 += h_cmplx_square(e2_re, e2_im);
                    s1_3 += h_cmplx_square(e3_re, e3_im);

                    s2_0 += h_cmplx_tesseract(e0_re, e0_im);
                    s2_1 += h_cmplx_tesseract(e1_re, e1_im);
                    s2_2 += h_cmplx_tesseract(e2_re, e2_im);
                    s2_3 += h_cmplx_tesseract(e3_re, e3_im);
                }

                S1[t_bar/N * D / 2 * F + f * D / 2 + feed4].x = s1_0;
                S1[t_bar/N * D / 2 * F + f * D / 2 + feed4].y = s1_1;
                S1[t_bar/N * D / 2 * F + f * D / 2 + feed4].z = s1_2;
                S1[t_bar/N * D / 2 * F + f * D / 2 + feed4].w = s1_3;

                S2[t_bar/N * D / 2 * F + f * D / 2 + feed4].x = s2_0;
                S2[t_bar/N * D / 2 * F + f * D / 2 + feed4].y = s2_1;
                S2[t_bar/N * D / 2 * F + f * D / 2 + feed4].z = s2_2;
                S2[t_bar/N * D / 2 * F + f * D / 2 + feed4].w = s2_3;
            }
        }
    }
}


float M_func(float mu) {
    return 0;
}

float V_func(float mu) {
    return 1;
}


void naive_mask(uint32_t *R, uint32_t *W, float4 *S1, float4 *S2, size_t N, size_t D, size_t T_bar, size_t F, float mu_min, float N_good_min, float sigma, float *SK, float *mean_SK, float *var_SK) {
    float N_good = 0;
    for (int w = 0; w < D * 2; w++) {
        N_good += (int) W[w];
    }
    printf("N_good = %f \n", N_good);

    if (N_good < N_good_min) {
        for (int r = 0; r < F * T_bar / 32; r++) {
            R[r] = 0;
        }
        printf("NOT ENOUGH N_GOOD \n");
        return;
    }

    float mu[D * 2 * F * T_bar];
    float S2_tilde[D * 2 * F * T_bar];
    for (int s = 0; s < D / 2 * F * T_bar; s++) {
        mu[s * 4 + 0] = S1[s].x / (float) N;
        mu[s * 4 + 1] = S1[s].y / (float) N;
        mu[s * 4 + 2] = S1[s].z / (float) N;
        mu[s * 4 + 3] = S1[s].w / (float) N;

        if (mu[s * 4 + 0] < mu_min) { S2_tilde[s * 4 + 0] = 0; } 
        else { S2_tilde[s * 4 + 0] = S2[s].x / (mu[s * 4 + 0] * mu[s * 4 + 0]); }
        if (mu[s * 4 + 1] < mu_min) { S2_tilde[s * 4 + 1] = 0; } 
        else { S2_tilde[s * 4 + 1] = S2[s].y / (mu[s * 4 + 1] * mu[s * 4 + 1]); }
        if (mu[s * 4 + 2] < mu_min) { S2_tilde[s * 4 + 2] = 0; } 
        else { S2_tilde[s * 4 + 2] = S2[s].z / (mu[s * 4 + 2] * mu[s * 4 + 2]); }
        if (mu[s * 4 + 3] < mu_min) { S2_tilde[s * 4 + 3] = 0; } 
        else { S2_tilde[s * 4 + 3] = S2[s].w / (mu[s * 4 + 3] * mu[s * 4 + 3]); }
    }
    printf("S1[0]=%f, mu[0]=%f, S2[0]=%f, S2_tilde[0]=%f\n", S1[0].x, mu[0], S2[0].x, S2_tilde[0]); 

    float sum;
    float mean_sum;
    float var_sum;
    
    float frac = (1 / N_good) * (((float) N + 1) / (float) (N - 1));
    float mean_frac = 1 + 1 / N_good;
    float var_frac = 4 / (h_square(N_good) * (float) N);

    // these can be single floats since values are used to compute R in the same loop
    float sk; 
    float mean_sk;
    float var_sk;

    for (int f = 0; f < F; f++) {    
        for (int t = 0; t < T_bar; t++) {
            sum = 0;
            mean_sum = 0;
            var_sum = 0;
            for (int pd = 0; pd < D * 2; pd++) {
                if (f == 0 & t == 0){ printf("sum=%f, w=%f, mu=%f, s2_tilde=%f, new=%f\n", sum, (float) W[pd], mu[t * F * D * 2 + f * D * 2 + pd], S2_tilde[t * F * D * 2 + f * D * 2 + pd], (float) W[pd] * ((S2_tilde[t * F * D * 2 + f * D * 2 + pd] / (float) N) - 1.)); }
                if (mu[t * F * D * 2 + f * D * 2 + pd] >= mu_min){
                    sum += (float) W[pd] * ((S2_tilde[t * F * D * 2 + f * D * 2 + pd] / (float) N) - 1.);
                    mean_sum += (float) W[pd] * M_func(mu[t * F * D * 2 + f * D * 2 + pd]);
                    var_sum += (float) W[pd] * V_func(mu[t * F * D * 2 + f * D * 2 + pd]);
                }
                // if (f == 0 & t == 0 & pd == 0){ printf("s2_tilde[0]=%f\n", S2_tilde[t * F * D * 2 + f * D * 2 + pd]); }
                // if (f == 0 & t == 0) { printf("w=%lu, sum=%f, mean_sum=%f, var_sum=%f\n", (unsigned long) W[pd], sum, mean_sum, var_sum); }
            }
            // printf("f=%d, t=%d, sum=%f, mean_sum=%f, var_sum=%f\n", f, t, sum, mean_sum, var_sum);
            sk = frac * sum;
            mean_sk = mean_frac * mean_sum;
            var_sk = var_frac * var_sum;

            if (f == 0 & t == 0) { 
            printf("at f=t=0, frac=%f, sum=%f, sk=%f, mean_sk=%f, var_sk=%f\n", frac, sum, sk, mean_sk, var_sk);
            }

            // R[f * T_bar + t] = ((abs(sk - mean_sk) <= (sigma * sqrt(var_sk))) ? true: false);
            if (abs(sk - mean_sk) <= (sigma * sqrt(var_sk))) { R[f * T_bar + t] = true; }
            else { R[f * T_bar + t] = false; }

            // temp
            SK[f * T_bar + t] = sk;
            mean_SK[f * T_bar + t] = mean_sk;
            var_SK[f * T_bar + t] = var_sk;
        }
    }
}


void test_downsample() {
    uint32_t *h_E, *d_E;
    float4 *h_S1, *h_S2, *h_S1_p, *h_S2_p;
    float4 *naive_S1, *naive_S2, *naive_S1_p, *naive_S2_p;
    float4 *d_S1, *d_S2, *d_S1_p, *d_S2_p;
    const size_t N = 2;//256;
    const size_t N_p = 4;//256 * 128;
    const size_t D = 64; // 64 or 512
    const size_t T = 32;//98304;
    const size_t F = 1;//50;

    h_E = (uint32_t*)malloc(sizeof(uint32_t) * D / 2 * F * T);
    h_S1 = (float4*)malloc(sizeof(float4) * D / 2 * F * (T/N));
    h_S2 = (float4*)malloc(sizeof(float4) * D / 2 * F * (T/N));
    h_S1_p = (float4*)malloc(sizeof(float4) * D / 2 * F * (T/N_p));
    h_S2_p = (float4*)malloc(sizeof(float4) * D / 2 * F * (T/N_p));

    naive_S1 = (float4*)malloc(sizeof(float4) * D / 2 * F * (T/N));
    naive_S2 = (float4*)malloc(sizeof(float4) * D / 2 * F * (T/N));
    naive_S1_p = (float4*)malloc(sizeof(float4) * D / 2 * F * (T/N_p));
    naive_S2_p = (float4*)malloc(sizeof(float4) * D / 2 * F * (T/N_p));

    gpuErrchk(hipMalloc((void**)&d_E, sizeof(uint32_t) * D / 2 * F * T));
    gpuErrchk(hipMalloc((void**)&d_S1, sizeof(float4) * D / 2 * F * (T/N)));
    gpuErrchk(hipMalloc((void**)&d_S2, sizeof(float4) * D / 2 * F * (T/N)));
    gpuErrchk(hipMalloc((void**)&d_S1_p, sizeof(float4) * D / 2 * F * (T/N_p)));
    gpuErrchk(hipMalloc((void**)&d_S2_p, sizeof(float4) * D / 2 * F * (T/N_p)));

    // TODO safe to assume D = 64 or 512 ONLY?
    dim3 blocks(F, D == 64 ? D / (32 * 2) : D / (32 * 4 * 2), T/N_p);
    dim3 threads(D == 64 ? 32 : 32 * 4);  // originally 2D/4. 2D bc dish and x- or y- polarisation pairs, 
                    // /4 bc 16 registers/thread, each holds 4 feeds. 16/4=4, one for each output array


    generate_random(h_E, D / 2 * F * T);

    // for (int t = 0; t < T; t++) {
    //     for (int f = 0; f < F; f++) {
    //         for (int feed4 = 0; feed4 < D / 2; feed4++) {
    //             uint32_t e = h_E[t * F * D / 2 + f * D / 2 + feed4];
    //             float e0_im = float((e >> 4) & 0xf);
    //             float e0_re = float(e & 0xf);
    //             float e1_re = float((e >> 8) & 0xf);
    //             float e1_im = float((e >> 12) & 0xf);
    //             float e2_re = float((e >> 16) & 0xf);
    //             float e2_im = float((e >> 20) & 0xf);
    //             float e3_re = float((e >> 24) & 0xf);
    //             float e3_im = float((e >> 28) & 0xf);

    //             printf("index %lu: %f + i%f, %f + i%f, %f + i%f, %f + i%f, ", t * F * D / 2 + f * D / 2 + feed4, e0_re, e0_im, e1_re, e1_im, e2_re, e2_im, e3_re, e3_im);
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");
    // }

    clock_t before_naive = clock();

    naive_downsample(h_E, naive_S1, naive_S2, naive_S1_p, naive_S2_p, N, N_p, D, T, F);

    double difference_naive = (double)(clock() - before_naive) / CLOCKS_PER_SEC;

    gpuErrchk(hipMemcpy(d_E, h_E, sizeof(uint32_t) * D / 2 * F * T, hipMemcpyHostToDevice));

    clock_t before = clock();

    downsample<<< blocks, threads >>>(d_E, d_S1, d_S2, d_S1_p, d_S2_p, N, N_p, D, T, F); 
    
    gpuErrchk(hipDeviceSynchronize());

    double difference = (double)(clock() - before) / CLOCKS_PER_SEC;


    gpuErrchk(hipMemcpy(h_S1, d_S1, sizeof(float4) * D / 2 * F * (T/N), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_S2, d_S2, sizeof(float4) * D / 2 * F * (T/N), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_S1_p, d_S1_p, sizeof(float4) * D / 2 * F * (T/N_p), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_S2_p, d_S2_p, sizeof(float4) * D / 2 * F * (T/N_p), hipMemcpyDeviceToHost));

    bool match = true;
    if (float4_arrays_equal(h_S1, naive_S1, D / 2 * F * (T/N)) == 0) {
        printf("S1 does not match \n");
        match = false;
    }
    if (float4_arrays_equal(h_S1_p, naive_S1_p, D / 2 * F * (T/N_p)) == 0) {
        printf("S1' does not match \n");
        match = false;
    }
    if (float4_arrays_equal(h_S2, naive_S2, D / 2 * F * (T/N)) == 0) {
        printf("S2 does not match \n");
        match = false;
    }
    if (float4_arrays_equal(h_S2_p, naive_S2_p, D / 2 * F * (T/N_p)) == 0) {
        printf("S2' does not match \n");
        match = false;
    }

    printf("Naive runtime: %f \n", difference_naive);
    printf("Kernel runtime: %f \n", difference);
    printf("Solution match: %d \n", match);
}


void test_mask() {
    // declare everything
    uint32_t *h_E, *d_E;
    uint32_t *h_R, *d_R, *h_W, *d_W;
    uint32_t *naive_R;
    float4 *h_S1, *h_S2, *h_S1_p, *h_S2_p, *d_S1, *d_S2, *d_S1_p, *d_S2_p;
    const size_t N = 2;//2; 265;
    const size_t N_p = 4;//4; 256 * 128;
    const size_t D = 64; // 64 or 512, needs to be multiple of 64
    const size_t T = 64; //64; 98304;
    const size_t T_bar = T / N;
    const size_t F = 1;
    const float sigma = 5;
    float *naive_SK, *naive_mean_SK, *naive_var_SK, *d_SK, *d_mean_SK, *d_var_SK, *h_SK, *h_mean_SK, *h_var_SK; // temp for testing

    // malloc arrays on host
    h_E = (uint32_t*)malloc(sizeof(uint32_t) * D / 2 * F * T);
    h_R = (uint32_t*)malloc(sizeof(uint32_t) * F * T_bar);
    h_W = (uint32_t*)malloc(sizeof(uint32_t) * D * 2);
    h_S1 = (float4*)malloc(sizeof(float4) * D / 2 * F * T_bar);
    h_S2 = (float4*)malloc(sizeof(float4) * D / 2 * F * T_bar);
    h_S1_p = (float4*)malloc(sizeof(float4) * D / 2 * F * (T/N_p));
    h_S2_p = (float4*)malloc(sizeof(float4) * D / 2 * F * (T/N_p));

    naive_R = (uint32_t*)malloc(sizeof(uint32_t) * F * T_bar);

    // temp for testing
    naive_SK = (float*)malloc(sizeof(float) * F * T/N);
    naive_mean_SK = (float*)malloc(sizeof(float) * F * T/N);
    naive_var_SK = (float*)malloc(sizeof(float) * F * T/N);
    h_SK = (float*)malloc(sizeof(float) * F * T/N);
    h_mean_SK = (float*)malloc(sizeof(float) * F * T/N);
    h_var_SK = (float*)malloc(sizeof(float) * F * T/N);

    // malloc arrays on device
    gpuErrchk(hipMalloc((void**)&d_E, sizeof(uint32_t) * D / 2 * F * T));
    gpuErrchk(hipMalloc((void**)&d_R, sizeof(uint32_t) * F * T_bar));
    gpuErrchk(hipMalloc((void**)&d_W, sizeof(uint32_t) * D * 2));
    gpuErrchk(hipMalloc((void**)&d_S1, sizeof(float4) * D / 2 * F * T_bar));
    gpuErrchk(hipMalloc((void**)&d_S2, sizeof(float4) * D / 2 * F * T_bar));
    gpuErrchk(hipMalloc((void**)&d_S1_p, sizeof(float4) * D / 2 * F * T_bar));
    gpuErrchk(hipMalloc((void**)&d_S2_p, sizeof(float4) * D / 2 * F * T_bar));

    // temp for testing
    gpuErrchk(hipMalloc((void**)&d_SK, sizeof(float) * F * T/N));
    gpuErrchk(hipMalloc((void**)&d_mean_SK, sizeof(float) * F * T/N));
    gpuErrchk(hipMalloc((void**)&d_var_SK, sizeof(float) * F * T/N));

    // generate fake data 
    generate_random_ones(h_W, D * 2);
    generate_noise_array(h_E, D / 2 * F * T);
    // generate_random_float4(h_S1, D / 2 * F * T_bar);
    // generate_random_float4(h_S2, D / 2 * F * T_bar);

    dim3 down_blocks(F, D == 64 ? D / (32 * 2) : D / (32 * 4 * 2), T/N_p);
    dim3 down_threads(D == 64 ? 32 : 32 * 4);  // originally 2D/4. 2D bc dish and x- or y- polarisation pairs, 
                    // /4 bc 16 registers/thread, each holds 4 feeds. 16/4=4, one for each output array
    gpuErrchk(hipMemcpy(d_E, h_E, sizeof(uint32_t) * D / 2 * F * T, hipMemcpyHostToDevice));

    downsample<<< down_blocks, down_threads >>>(d_E, d_S1, d_S2, d_S1_p, d_S2_p, N, N_p, D, T, F); 

    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(h_S1, d_S1, sizeof(float4) * D / 2 * F * T_bar, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_S2, d_S2, sizeof(float4) * D / 2 * F * T_bar, hipMemcpyDeviceToHost));

    // =================== MASK =====================

    // start the timer for naive solution
    clock_t before_naive = clock();

    // run naive solution
    naive_mask(naive_R, h_W, h_S1, h_S2, N, D, T_bar, F, 1, 1, sigma, naive_SK, naive_mean_SK, naive_var_SK);

    // print bunch of inputs and outputs TODO delete
    // printf("W \n");
    // for (int pd = 0; pd < D * 2; pd++) {
    //     printf("%d, ", h_W[pd]);
    // }
    // printf("\n");

    // printf("S1 \n");
    // for (int s = 0; s < D / 2; s++) { // s < D / 2 * F * T_bar
    //     printf("%f,%f,%f,%f,", h_S1[s].x, h_S1[s].y, h_S1[s].z, h_S1[s].w);
    // }
    // printf("\n");

    // printf("S2 \n");
    // for (int s = 0; s < D / 2; s++) { // s < D / 2 * F * T_bar
    //     printf("%f, %f, %f, %f, ", h_S2[s].x, h_S2[s].y, h_S2[s].z, h_S2[s].w);
    // }
    // printf("\n");

    // printf("S1[0] = %f, S2[0] = %f\n", h_S1[0].x, h_S2[0].x);

    // printf("R\n");
    // for (int f = 0; f < F; f++) {
    //     for (int t = 0; t < T_bar; t++) {
    //         printf("%d ", naive_R[f * T_bar + t]);
    //     }
    //     printf("\n");
    // }

    // end naive timer
    double difference_naive = (double)(clock() - before_naive) / CLOCKS_PER_SEC;

    // copy input host to device
    gpuErrchk(hipMemcpy(d_W, h_W, sizeof(uint32_t) * D * 2, hipMemcpyHostToDevice));
    // gpuErrchk(hipMemcpy(d_S1, h_S1, sizeof(float4) * D / 2 * F * T_bar, hipMemcpyHostToDevice));
    // gpuErrchk(hipMemcpy(d_S2, h_S2, sizeof(float4) * D / 2 * F * T_bar, hipMemcpyHostToDevice));

    // define num blocks and threads
    dim3 blocks(F, T_bar / 32);
    dim3 threads(32 * 32); // 32 warps, each 32 threads (one coarse time index t_bar computed on each warp)

    printf("blocks: %d, %d, %d; threads: %d, %d, %d \n", blocks.x, blocks.y, blocks.z, threads.x, threads.y, threads.z);
    // time and run parallel solution
    clock_t before = clock();

    mask<<< blocks, threads >>>(d_R, d_W, (float*) d_S1, (float*) d_S2, N, D, T_bar, F, 1, 1, sigma, d_SK, d_mean_SK, d_var_SK); 
    printf("peek: %d\n", hipPeekAtLastError());
   
    gpuErrchk(hipDeviceSynchronize());
    printf("peek: %d\n", hipPeekAtLastError());

    double difference = (double)(clock() - before) / CLOCKS_PER_SEC;

    // copy output device to host
    gpuErrchk(hipMemcpy(h_R, d_R, sizeof(uint32_t) * F * T_bar, hipMemcpyDeviceToHost));
    // test for testing
    gpuErrchk(hipMemcpy(h_SK, d_SK, sizeof(float) * F * T_bar, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_mean_SK, d_mean_SK, sizeof(float) * F * T_bar, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_var_SK, d_var_SK, sizeof(float) * F * T_bar, hipMemcpyDeviceToHost));

    // printf("GPU R\n");
    // for (int f = 0; f < F; f++) {
    //     for (int t = 0; t < T_bar; t++) {
    //         printf("%d ", h_R[f * T_bar + t]);
    //     }
    //     printf("\n");
    // }

    // check if solutions match
    bool match = true;
    if (uint32_arrays_equal(h_R, naive_R, F * T_bar) == 0) {
        printf("R does not match \n");
        match = false;
    }

    if (float_arrays_equal(h_SK, naive_SK, F * T_bar) == 0) {
        printf("SK does not match \n");
        match = false;
    }

    if (float_arrays_equal(h_mean_SK, naive_mean_SK, F * T_bar) == 0) {
        printf("mean_SK does not match \n");
        match = false;
    }

    if (float_arrays_equal(h_var_SK, naive_var_SK, F * T_bar) == 0) {
        printf("var_SK does not match \n");
        match = false;
    }

    // print results
    printf("Naive runtime: %f \n", difference_naive);
    printf("Kernel runtime: %f \n", difference);
    printf("Solution match: %d \n", match);
}


int main() {
    test_mask();
}
